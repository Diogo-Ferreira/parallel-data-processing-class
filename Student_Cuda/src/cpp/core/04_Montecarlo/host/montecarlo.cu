#include "hip/hip_runtime.h"
#include "montecarlo.h"

#include <iostream>

#include "Device.h"
#include <hiprand/hiprand_kernel.h>
using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void montecarlo(hiprandState* tabDevGenerator, int* ptrDevGMRes, long n, float m);
extern __global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const Grid& grid, float xMin, float xMax, float m, int n) :
	 n(n), xMin(xMin), xMax(xMax), m(m)
    {
    this->sizeOctet = sizeof(float); // octet
    this->randSizeOctet = sizeof(hiprandState);


    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}

	int nbThread = grid.threadCounts();
	cout << "nb threads" << nbThread << endl;
	hiprandState* ptrDevGenerator=nullptr;

	Device::malloc(&this->piHatDev, sizeOctet);
	Device::memclear(this->piHatDev, sizeOctet);

	Device::malloc(&this->tabDevGenerator, randSizeOctet*nbThread);
	Device::memclear(this->tabDevGenerator, randSizeOctet*nbThread);
	this->nbArrows = n/nbThread;

	sizeOctetSM = grid.db.x * sizeof(float);

	//Device::memcpyHToD(this->piHatDev, this->piHat, sizeOctet);
    }

Montecarlo::~Montecarlo(void)
    {
    //MM (device free)
	{
	Device::free(piHatDev);

	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug, facultatif
	}
    }
/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Montecarlo::run()
    {
    Device::lastCudaError("Slice (before)"); // temp debug

    setup_kernel_rand<<<dg,db>>>(this->tabDevGenerator, Device::getDeviceId());
    cout << "nb arrows" << nbArrows << endl;
    montecarlo<<<dg,db, sizeOctetSM>>>(this->tabDevGenerator, piHatDev, nbArrows, m); // assynchrone
    Device::lastCudaError("addVecteur (after)"); // temp debug

    //Device::synchronize(); // Temp,debug, only for printf in  GPU

    // MM (Device -> Host)
	{
	Device::memcpyDToH(&arrowsBelow, piHatDev, sizeOctet); // barriere synchronisation implicite
	}

	cout << "hello " << arrowsBelow << endl;

	//this->piHat = arrowsBelow/nbArrows*((xMax-xMin)*m);
	this->piHat = 2.0*(double)arrowsBelow/(double)n*(xMax-xMin)*m;
    }

float Montecarlo::getResult()
    {
	return piHat;
    }

int Montecarlo::getCountArrows()
    {
	return arrowsBelow;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
