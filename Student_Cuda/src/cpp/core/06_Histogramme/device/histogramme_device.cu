#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include <stdio.h>

__global__ void histogramd(int* ptrTabData, int tabSize, int *ptrDevResult);
static __device__ void reductionIntraThread(int *ptrTabData, int tabSize, int *tabSM);
__device__ void reductionInterBlock(int* TAB_SM, int* ptrGM);

__global__ void histogramd(int* ptrTabData, int tabSize, int *ptrDevResult)
    {
    extern __shared__ int tabSM[];


    reductionIntraThread(ptrTabData, tabSize, tabSM);
    __syncthreads();
    reductionInterBlock(tabSM, ptrDevResult);

    }
__device__ void reductionIntraThread(int *ptrTabData, int tabSize, int *tabSM)
    {
    const int NB_THREAD = Indice2D::nbThread();
    const int TID_LOCAL = Indice2D::tidLocal();
    const int TID = Indice2D::tid();

    int s = TID;

    while (s < tabSize)
	{
	atomicAdd(&tabSM[ptrTabData[s]], 1);
	s += NB_THREAD;
	}
    }

__device__ void reductionInterBlock(int* tabSM, int* tabGM)
    {
    if (Indice2D::tidLocal() == 0)
    	    {
    	    for (int i = 0; i < 256; i++)
    		{
    		atomicAdd(&tabGM[i], tabSM[i]);
    		}
    	    }
    }
