#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void slice(int n, float* ptrDevGMRes);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(float* tabSM, int nbSlice);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void slice(int nbSlice, float* ptrDevGMRes)
    {
    extern __shared__ float tabSM[];
    reductionIntraThread(tabSM, nbSlice);
    __syncthreads();
    reductionADD<float>(tabSM, ptrDevGMRes);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(float* tabSM, int nbSlice)
    {
    //pattern d'entrelacement
    const int NB_THREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();
    const float DX=1.0f/(float)nbSlice;

    float localSum = 0;
    int s = TID;
    while (s < nbSlice)
	{
	float xs = s*DX;
	localSum += 4.0f / (1.0f + xs * xs);
	//localSum++;
	s += NB_THREAD;
	}
    tabSM[TID_LOCAL] = localSum/(float)nbSlice;
    }


/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

