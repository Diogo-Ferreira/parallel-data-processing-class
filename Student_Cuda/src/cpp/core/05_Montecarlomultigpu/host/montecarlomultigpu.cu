#include "hip/hip_runtime.h"
#include "../../05_Montecarlomultigpu/host/montecarlomultigpu.h"

#include <iostream>

#include "Device.h"
#include <hiprand/hiprand_kernel.h>
#include "montecarlo.h"
using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Montecarlomultigpu::Montecarlomultigpu(const Grid& grid, float xMin, float xMax, float m, int n) :
	 n(n), xMin(xMin), xMax(xMax), m(m)
    {
	this->grid = grid;
	//Device::memcpyHToD(this->piHatDev, this->piHat, sizeOctet);
    }

Montecarlomultigpu::~Montecarlomultigpu(void)
    {
    //MM (device free)
	{
	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug, facultatif
	}
    }
/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Montecarlomultigpu::run()
    {
	int nbDevice=Device::getDeviceCount() ;
	int nbFlechetteGPU=n/nbDevice;
	int sumTotal=0 ;
	cout << "Nombre de device " << nbDevice << endl;
	#pragma omp parallel for reduction(+:sumTotal)
	for (int idDevice=0; idDevice< nbDevice ; idDevice ++)
	    {
	    hipSetDevice (idDevice) ; // idDevice nintervient plus ensuite
	    Montecarlo montecarlo(grid, xMin, xMax, m, nbFlechetteGPU) ;// sur le device courant !
	    montecarlo.run() ; // sur le device courant !
	    sumTotal += montecarlo.getCountArrows();
	    }

	cout << "Somme du total : " << sumTotal << endl;
	// Finalisation mathématique coté host
	//piHat = 2.0f*(float)sumTotal/(float)n*(fabsf(xMax-xMin)*m);
	this->piHat = 2.0*(double)sumTotal/(double)n*((double)xMax-(double)xMin)*(double)m;


    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
