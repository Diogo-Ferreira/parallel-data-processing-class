#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"
#include "histogram.h"


using std::cout;
using std::endl;

const int SIZE_TAB_SM = 256;

extern __global__ void histogramd(int* ptrTabData, int n, int *ptrDevResult);

Histogram::Histogram(const Grid& grid, int tabSize) :
		tabSize(tabSize)
{
	this->ptrTabResult = new int[SIZE_TAB_SM];

	this->dataCreator = new DataCreator(tabSize);
	this->ptrTabData = dataCreator->getTabData();
	this->n = dataCreator->getLength();

	this->sizeOctetResult = sizeof(int) * SIZE_TAB_SM; // octet
	this->sizeOctetData = sizeof(int) * this->n;

	Device::malloc(&this->ptrDataDev, sizeOctetData);
	Device::memclear(this->ptrDataDev, sizeOctetData);

	Device::malloc(&this->ptrResultDev, sizeOctetResult);
	Device::memclear(this->ptrResultDev, sizeOctetResult);

	Device::memcpyHToD(this->ptrDataDev, this->ptrTabData, this->sizeOctetData);


	this->dg = grid.dg;
	this->db = grid.db;




}

Histogram::~Histogram(void)
{
	Device::free(ptrResultDev);
	Device::free(ptrDataDev);
}

int* Histogram::run()
{
	histogramd<<<dg,db,sizeOctetResult>>>(this->ptrDataDev, this->n, ptrResultDev); // asynchrone
	Device::synchronize();

	Device::memcpyDToH(ptrTabResult, ptrResultDev, sizeOctetResult);

	return ptrTabResult;
}
