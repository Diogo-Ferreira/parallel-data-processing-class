#include "hip/hip_runtime.h"
#include "slice.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void slice(int n, float* GM);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Slice::Slice(const Grid& grid, int n) :
	 n(n)
    {
    this->sizeOctet = sizeof(float); // octet



    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
	Device::malloc(&this->piHatDev, sizeOctet);
	Device::memclear(this->piHatDev, sizeOctet);

	sizeOctetSM = grid.db.x * sizeof(float);

	//Device::memcpyHToD(this->piHatDev, this->piHat, sizeOctet);
    }

Slice::~Slice(void)
    {
    //MM (device free)
	{
	Device::free(piHatDev);

	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Slice::run()
    {
    Device::lastCudaError("Slice (before)"); // temp debug
    slice<<<dg,db, sizeOctetSM>>>(n, piHatDev); // assynchrone
    Device::lastCudaError("addVecteur (after)"); // temp debug

    //Device::synchronize(); // Temp,debug, only for printf in  GPU

    // MM (Device -> Host)
	{
	Device::memcpyDToH(&piHat, piHatDev, sizeOctet); // barriere synchronisation implicite
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
