#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>

#include "DomaineMath_GPU.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandel(uchar4* ptrDevPixels,uint w, uint h,float t,DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float dt,const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Cuda_RGBA_uchar4",domaineMath), variateurAnimation(Interval<float>(20, 120), dt)
    {
    assert(w == h); // specific rippling

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    }

Mandelbrot::~Mandelbrot()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rippling rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    float t=variateurAnimation.get();
    mandel<<<dg,db>>> (ptrDevPixels,w,h,t,domaineMath);

    Device::lastCudaError("rippling rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::animationStep()
    {
    t += variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

