#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "DomaineMath_GPU.h"
#include "MandelMath.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandel(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandel(uchar4* ptrDevPixels, uint w, uint h, float t,DomaineMath domaineMath)
    {
    MandelMath mandelMath = MandelMath(w,h);

    const int TID = Indice2D::tid();
    const int NB_THREADS = Indice2D::nbThread();
    const int WH = w * h;
    int i;
    int j;
    int s = TID;
    while(s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	double x;
	double y;
	domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)
	mandelMath.colorXY(&ptrDevPixels[s],x, y, t);
	s +=NB_THREADS;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

