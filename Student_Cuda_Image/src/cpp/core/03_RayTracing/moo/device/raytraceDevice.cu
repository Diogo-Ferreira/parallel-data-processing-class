#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "../length_cm.h"

#include "DomaineMath_GPU.h"
#include "RaytraceMath.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void raytraceGM(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSpheres, uint w, uint h, float t);
__host__ void uploadGPU(Sphere* tabValue);
__global__ void raytraceCM(uchar4* ptrDevPixels, uint w, uint h, float t);
__device__ void work(uchar4* ptrDevPixels,Sphere* ptrDevSphere, int n, uint w, uint h, float t);
__global__ void rayTracingSM(uchar4* ptrDevPixels, uint w, uint h, float dt, Sphere* ptrDevTabSphere);
__device__ void copyGMtoSM(Sphere* ptrDevTabSphere, Sphere* tab_SM);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void raytraceGM(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSpheres, uint w, uint h, float t)
    {
    work(ptrDevPixels, ptrDevTabSphere, nbSpheres, w, h, t);
    }

// Déclaration Constante globale
__constant__ Sphere SPHERE_CM[LENGTH_CM];
/**
 * call once by the host
 */
__host__ void uploadGPU(Sphere* tabValue)
    {
    size_t size = LENGTH_CM * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(SPHERE_CM), tabValue, size, offset, hipMemcpyHostToDevice));
    }

__device__ void copyGMtoSM(Sphere* ptrDevTabSphere, Sphere* tab_SM)
    {
    const int TID_LOCAL = Indice2D::tidLocal();
    const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();
    int s = TID_LOCAL;
    int NB_SPHERE = LENGTH_CM;

    while (s < NB_SPHERE)
	{
	tab_SM[s] = ptrDevTabSphere[s];
	s += NB_THREAD_LOCAL;
	}
    }

__global__ void raytraceCM(uchar4* ptrDevPixels, uint w, uint h, float t)
    {

    work(ptrDevPixels, SPHERE_CM, LENGTH_CM, w, h, t);
    }

__global__ void rayTracingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere)
    {
    extern __shared__ Sphere tab_SM[];
    copyGMtoSM(ptrDevTabSphere, tab_SM);

    __syncthreads();
    work(ptrDevPixels, tab_SM, LENGTH_CM, w, h, t);
    }
__device__ void work(uchar4* ptrDevPixels,Sphere* ptrDevSphere, int n, uint w, uint h, float t)
{
    RaytraceMath raytraceMath = RaytraceMath(w, h, ptrDevSphere, n);

    const int TID = Indice2D::tid();
    const int NB_THREADS = Indice2D::nbThread();
    const int WH = w * h;
    int i;
    int j;
    int s = TID;

    while (s < WH)
    {
    IndiceTools::toIJ(s, w, &i, &j);
    raytraceMath.colorIJ(&ptrDevPixels[s], i, j, t);
    s += NB_THREADS;
    }
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

