#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <Indice1D.h>
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);
__global__ void montecarlo(hiprandState* tabDevGenerator, int* ptrDevGMRes, long n, float m);

/*--------------------------------------*\sliisOkisOkcei
 * sOk
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM, int n, hiprandState* tabDevGenerator, float m);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void montecarlo(hiprandState* tabDevGenerator, int* ptrDevGMRes, long n, float m)
    {
    extern __shared__ int tabSM[];
    reductionIntraThread(tabSM, n, tabDevGenerator, m);
    __syncthreads();
    reductionADD<int>(tabSM, ptrDevGMRes);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(int* tabSM, int n, hiprandState* tabDevGenerator, float m)
    {
    //pattern d'entrelacement
    const int NB_THREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();

    float inCount = 0;
    hiprandState localGenerator = tabDevGenerator [TID];
    float xAlea;
    float yAlea;
    float fx;
    for (long i = 1; i <= n; i++)
	{

	xAlea = hiprand_uniform(&localGenerator);
	yAlea = hiprand_uniform(&localGenerator)*m;

	//fx = 4.0f / (1.0f + xAlea * xAlea);
	fx = sqrtf(1-(xAlea * xAlea));

	if(yAlea < fx){
	    inCount++;
	}
	}
    tabSM[TID_LOCAL] = inCount;
    }


// Each thread gets same seed, a different sequence number
// no offset
__global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId)
    {
// Customisation du generator:
// Proposition, au lecteur de faire mieux !
// Contrainte : Doit etre différent d'un GPU à l'autre
// Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGenerator[TID]);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

