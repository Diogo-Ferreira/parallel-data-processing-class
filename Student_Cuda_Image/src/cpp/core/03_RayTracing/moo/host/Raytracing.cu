#include "hip/hip_runtime.h"
#include "Raytracing.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>

#include "length_cm.h"

#include "SphereCreator.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytraceGM(uchar4* ptrDevPixels,Sphere* ptrDevTabSphere,int nbSpheres,uint w, uint h,float t);
extern __host__ void uploadGPU(Sphere* tabValue);
extern __global__ void raytraceCM(uchar4* ptrDevPixels, uint w, uint h, float t);
extern __global__ void rayTracingSM(uchar4* ptrDevPixels, uint w, uint h, float dt, Sphere* ptrDevTabSphere);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Raytracing::Raytracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4"), variateurAnimation(Interval<float>(0, 120),dt)
    {
    assert(w == h); // specific rippling

    // Inputs
    this->dt = dt;
    // Tools
    this->t = 0; // protected dans Animable
    this->nbSphere = LENGTH_CM;

    SphereCreator shereCreator(this->nbSphere, w, h); // sur la pile
    Sphere* ptrTabSphere = shereCreator.getTabSphere();

    this->size_octets = sizeof(Sphere) * this->nbSphere;

    // transfert to GM
    toGM(ptrTabSphere); // a implemneter

    // transfert to CM
    fillCM(ptrTabSphere);

    }

Raytracing::~Raytracing()
    {
    Device::free(this->ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

void Raytracing::toGM(Sphere* ptrSphere)
    {

    Device::malloc(&this->ptrDevTabSphere, size_octets);

    Device::memcpyHToD(this->ptrDevTabSphere, ptrSphere, size_octets);
    }

__host__ void Raytracing::fillCM(Sphere* ptrSphere)
    {
// Appelle le service d’upload coté device
    uploadGPU(ptrSphere);
    }

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Raytracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rippling rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    t=variateurAnimation.get();
    //raytrace<<<dg,db>>> (ptrDevPixels,this->ptrDevTabSphere,this->nbSphere,w,h,t);
    //raytrace_cm<<<dg,db>>> (ptrDevPixels,w,h,t);
    static int i = 0;

    if (i % 3 == 0)
    {
    raytraceGM<<<dg,db>>>(ptrDevPixels,this->ptrDevTabSphere,this->nbSphere,w,h,t);
    }
    else if (i % 3 == 1)
    {
    raytraceCM<<<dg,db>>>(ptrDevPixels, w, h, t);
    }
    else if (i % 3 == 2)
    {
    rayTracingSM<<<dg,db,size_octets>>>(ptrDevPixels, w, h, t, ptrDevTabSphere);
    }
    i++;
    Device::lastCudaError("rippling rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Raytracing::animationStep()
    {
    t += variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

